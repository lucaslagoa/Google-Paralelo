
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <pthread.h>




typedef struct {
	int n_vector;
	int n_buffer;
	int *d_buffer;
	int *d2_buffer;
	int id;
} thread_arg_t;


pthread_mutex_t the_mutex;
pthread_cond_t condc, condp;
int aux = 0;

__global__ void kernel_create(int *data){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	//int i;
	
	data[ix] = data[ix];
	__syncthreads();
}


void call_consumer(int n_buffer, int *data);


void *launch_kernel_create(void* _arg){
	thread_arg_t *arg = (thread_arg_t*)_arg;
	int n_vector = arg->n_vector;
	int n_buffer = arg->n_buffer;
	
	int *d_buffer = arg->d_buffer;
	int *d2_buffer = arg->d2_buffer;
	hipStream_t stream_0;

	int i;
	int sum;
	int index;
	int cont;
	int *vector;
	int *buffer;

	dim3 grid, block;
	block.x = 1024;
	grid.x = (n_buffer + block.x - 1) / block.x;
	
	hipSetDevice(0);


	vector = (int*) malloc(sizeof(int)*n_vector);
	buffer = (int*) malloc(sizeof(int)*n_buffer);
	for(i=0;i<n_vector;i++){
		vector[i] = i+1;
	}
	
	index = 0;
	cont = 0;
	sum = 0;
	while(index<n_vector){

		//lockar mutex 2
		pthread_mutex_lock(&the_mutex);
		while(aux != 0) pthread_cond_wait(&condp, &the_mutex);

		for(i=0;i<(n_buffer);i++){
			buffer[i] = vector[i + index]; 
			sum += buffer[i]*2;
		}
		index = index + n_buffer;
		hipMemcpy(d_buffer,buffer,n_buffer*sizeof(int),hipMemcpyHostToDevice);
		kernel_create<<<grid,block>>>(d_buffer);
		hipMemcpyPeerAsync(d2_buffer,1,d_buffer,0,n_buffer*sizeof(int),stream_0);
		cont++;
		aux = 1;
		

		
		//liberar mutex 2
		pthread_cond_signal(&condc);
		pthread_mutex_unlock(&the_mutex);

	}

	printf("Soma total na CPU: %d Cont: %d\n",sum,cont-1);
	
	hipFree(d_buffer);

	return 0;
}

void *launch_kernel_modify(void* _arg){
	thread_arg_t *arg = (thread_arg_t*)_arg;
	int n_vector = arg->n_vector;
	int n_buffer = arg->n_buffer;
	int *d2_buffer = arg->d2_buffer;
	int *buffer2;
	int acessivel = 0;
	int index,i,j;
	int sum = 0;
	
	buffer2 = (int*) malloc(sizeof(int)*n_buffer);


	dim3 grid, block;
	block.x = 1024;
	grid.x = (n_buffer + block.x - 1) / block.x;


	hipDeviceCanAccessPeer(&acessivel,1,0);

	if(acessivel){
		int d_sum;
		hipSetDevice(1);
		hipDeviceEnablePeerAccess(0,0);
		index = n_vector/n_buffer;
		for(i=0;i<index;i++){
			//lockar mutex 1
			pthread_mutex_lock(&the_mutex);
			while(aux == 0) pthread_cond_wait(&condc, &the_mutex);


			hipSetDevice(1);			

			//kernel_modify<<<grid,block>>>(d2_buffer);
			call_consumer(n_buffer,d2_buffer);
			hipMemcpy(buffer2,d2_buffer,sizeof(int)*n_buffer,hipMemcpyDeviceToHost);
			for(j=0;j<n_buffer;j++){
				//printf("(%d,%d) ",j,buffer2[j]);
				sum+=buffer2[j];
			}
			aux = 0;
			//liberar mutex 1
			pthread_cond_signal(&condp);
			pthread_mutex_unlock(&the_mutex);
			
		}
		
		hipFree(&d_sum);
		hipFree(d2_buffer);
		printf("Somatorio na GPU: %d\n",sum);
		
	}


	return 0;

}



int main(void){


	int i;
	int gpu1 = 0;
	int gpu2 = 1;
	int n_buffer = 10;
	int n_vector = 1000;
	int *d_buffer;
	int *d2_buffer;




	pthread_t thread[2];
	thread_arg_t args[2];

	pthread_mutex_init(&the_mutex, NULL);	
	pthread_cond_init(&condc, NULL);		/* Initialize consumer condition variable */
  	pthread_cond_init(&condp, NULL);
	
	hipSetDevice(gpu1);
	hipMalloc(&d_buffer,sizeof(int)*n_buffer);

	hipSetDevice(gpu2);

	hipMalloc(&d2_buffer,sizeof(int)*n_buffer);

	
	
	hipSetDevice(gpu1);
	args[0].id = 0;
	args[0].d_buffer = d_buffer;
	args[0].d2_buffer = d2_buffer;
	args[0].n_vector = n_vector;
	args[0].n_buffer = n_buffer;
	pthread_create(&(thread[0]), NULL, launch_kernel_create, &(args[0]));
	
	
	hipSetDevice(gpu2);
	args[1].id = 0;
	args[1].n_buffer = n_buffer;
	args[1].n_vector = n_vector;
	args[1].d2_buffer = d2_buffer;
	pthread_create(&(thread[1]), NULL, launch_kernel_modify, &(args[1]));
	

	for(i=0;i<2;i++){
		pthread_join(thread[i],NULL);
	}


	pthread_mutex_destroy(&the_mutex);	/* Free up the_mutex */
	pthread_cond_destroy(&condc);		/* Free up consumer condition variable */
  	pthread_cond_destroy(&condp);	


	hipSetDevice(1);
	hipDeviceDisablePeerAccess(0);

	return 0;


}
