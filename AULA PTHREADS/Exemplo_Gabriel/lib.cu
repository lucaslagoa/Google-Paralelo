
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel_modify(int *data){
	int ix = blockIdx.x*blockDim.x + threadIdx.x;
	//int i;
	
	data[ix] = data[ix]*2;
	
	__syncthreads();
}


void call_consumer(int n_buffer, int *data){

	dim3 grid, block;
	block.x = 1024;
	grid.x = (n_buffer + block.x - 1) / block.x;

	kernel_modify<<<grid,block>>>(data);
	hipDeviceSynchronize();


}
